#include <string>
#include <iostream>
#include <stdint.h>
#include <unistd.h>

#include "inc/constants.h"
#include "inc/RBSP_structs.h"
#include "inc/types.h"
#include "inc/classes.h"

using std::cout;
using std::endl;
using std::string;

// create global variables
H264parser parser           = H264parser( );
hipDevice_t dev;
hipCtx_t* pCtx             = new hipCtx_t;
CUvideodecoder* pDecoder    = new CUvideodecoder;
CUVIDDECODECREATEINFO *pdci = new CUVIDDECODECREATEINFO;

// this function is run on data from one frame of video
// essentially, this is the processing callback
int frame_handler( uint8_t* start, uint32_t length )
{
	std::cerr << "." << std::flush;
	
	parser.parseFrame( start, length );
	
	cuvidDecodePicture( *pDecoder, parser.cuvidPicParams );
	
	int mapIdx = parser.idx( ) - 4;
	if( mapIdx >= 0 )
		mapSurface( mapIdx, parser.getProcParams( ) );

	return 0;
}

int main( int argc, char** argv )
{
	// create a V4L2 stream object
	V4L2stream stream = V4L2stream( );
	stream.init( );

	// CUDA code below is ugly and needs to be abstracted
	// create context to appease cuda runtime
	hipSetDevice( 0 );
	hipGetDevice( &dev );

	// fill video decoder creation struct
	pdci->ulWidth             = CODED_WIDTH;
	pdci->ulHeight            = CODED_HEIGHT;
	pdci->ulNumDecodeSurfaces = 15;
	pdci->CodecType           = CUVID_CODEC;
	pdci->ChromaFormat        = CUVID_CHROMA;
	pdci->ulCreationFlags     = CUVID_FLAGS;
	pdci->display_area.left   = 0;
	pdci->display_area.top    = 0;
	pdci->display_area.right  = TARGET_WIDTH;
	pdci->display_area.bottom = TARGET_HEIGHT;
	pdci->OutputFormat        = CUVID_OUT_FORMAT;
	pdci->DeinterlaceMode     = CUVID_DEINTERLACE;
	pdci->ulTargetWidth       = TARGET_WIDTH;
	pdci->ulTargetHeight      = TARGET_HEIGHT;
	pdci->ulNumOutputSurfaces = 8;
	pdci->vidLock             = NULL;
	pdci->target_rect.left    = 0;
	pdci->target_rect.top     = 0;
	pdci->target_rect.right   = TARGET_WIDTH;
	pdci->target_rect.bottom  = TARGET_HEIGHT;

	cuvidCreateDecoder( pDecoder, pdci );

	stream.on( );
	for( int i = 0; i < 1200; ++i) // "process" 1200 frames (40 seconds)
		stream.getFrame( &frame_handler );
	stream.off( );

	return 0;
}
